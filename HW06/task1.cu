#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include "matmul.cuh"

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " n threads_per_block" << std::endl;
        return -1;
    }

    size_t n = std::stoi(argv[1]);
    unsigned int threads_per_block = std::stoi(argv[2]);
    size_t size = n * n * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize matrices with random values between [-1, 1]
    srand(time(0));
    for (size_t i = 0; i < n * n; ++i) {
        h_A[i] = (float(rand()) / RAND_MAX) * 2 - 1;
        h_B[i] = (float(rand()) / RAND_MAX) * 2 - 1;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Record time using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul(d_A, d_B, d_C, n, threads_per_block);
    hipEventRecord(stop);

    // Wait for the event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the last element of the resulting matrix and execution time
    std::cout << "Last element: " << h_C[n * n - 1] << std::endl;
    std::cout << "Time taken: " << milliseconds << " ms" << std::endl;

    // Free host and device memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
