#include "matmul.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>


template <typename T>
void initialize_matrix(std::vector<T> &matrix, unsigned int n) {
    for (auto &val : matrix)
        val = static_cast<T>(rand() % 100);
}

int main(int argc, char **argv) {
    if (argc != 3) {
        std::cerr << "Usage: ./task1 n block_dim\n";
        return -1;
    }

    unsigned int n = atoi(argv[1]);
    unsigned int block_dim = atoi(argv[2]);


    std::vector<int> h_A_int(n * n), h_B_int(n * n), h_C_int(n * n);
    std::vector<float> h_A_float(n * n), h_B_float(n * n), h_C_float(n * n);
    std::vector<double> h_A_double(n * n), h_B_double(n * n), h_C_double(n * n);

    initialize_matrix(h_A_int, n);
    initialize_matrix(h_B_int, n);
    initialize_matrix(h_A_float, n);
    initialize_matrix(h_B_float, n);
    initialize_matrix(h_A_double, n);
    initialize_matrix(h_B_double, n);


    int *d_A_int, *d_B_int, *d_C_int;
    float *d_A_float, *d_B_float, *d_C_float;
    double *d_A_double, *d_B_double, *d_C_double;

    hipMalloc(&d_A_int, n * n * sizeof(int));
    hipMalloc(&d_B_int, n * n * sizeof(int));
    hipMalloc(&d_C_int, n * n * sizeof(int));
    hipMalloc(&d_A_float, n * n * sizeof(float));
    hipMalloc(&d_B_float, n * n * sizeof(float));
    hipMalloc(&d_C_float, n * n * sizeof(float));
    hipMalloc(&d_A_double, n * n * sizeof(double));
    hipMalloc(&d_B_double, n * n * sizeof(double));
    hipMalloc(&d_C_double, n * n * sizeof(double));


    hipMemcpy(d_A_int, h_A_int.data(), n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B_int, h_B_int.data(), n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A_float, h_A_float.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B_float, h_B_float.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A_double, h_A_double.data(), n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B_double, h_B_double.data(), n * n * sizeof(double), hipMemcpyHostToDevice);


    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    std::cout << "Testing matmul_1 (int):" << std::endl;
    hipEventRecord(start1);
    matmul_1(d_A_int, d_B_int, d_C_int, n, block_dim);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float time_1 = 0;
    hipEventElapsedTime(&time_1, start1, stop1);
    hipMemcpy(h_C_int.data(), d_C_int, n * n * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "First element: " << h_C_int[0] << std::endl;
    std::cout << "Last element: " << h_C_int[n * n - 1] << std::endl;
    std::cout << "Time taken: " << time_1 << " ms" << std::endl;
    hipEventDestroy(start1);
    hipEventDestroy(stop1);

    //matmul_2
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    std::cout << "Testing matmul_2 (float):" << std::endl;
    hipEventRecord(start2);
    matmul_2(d_A_float, d_B_float, d_C_float, n, block_dim);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    float time_2 = 0;
    hipEventElapsedTime(&time_2, start2, stop2);
    hipMemcpy(h_C_float.data(), d_C_float, n * n * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "First element: " << h_C_float[0] << std::endl;
    std::cout << "Last element: " << h_C_float[n * n - 1] << std::endl;
    std::cout << "Time taken: " << time_2 << " ms" << std::endl;
    hipEventDestroy(start2);
    hipEventDestroy(stop2);

    //matmul_3
    hipEvent_t start3, stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    std::cout << "Testing matmul_3 (double):" << std::endl;
    hipEventRecord(start3);
    matmul_3(d_A_double, d_B_double, d_C_double, n, block_dim);
    hipEventRecord(stop3);
    hipEventSynchronize(stop3);
    float time_3 = 0;
    hipEventElapsedTime(&time_3, start3, stop3);
    hipMemcpy(h_C_double.data(), d_C_double, n * n * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "First element: " << h_C_double[0] << std::endl;
    std::cout << "Last element: " << h_C_double[n * n - 1] << std::endl;
    std::cout << "Time taken: " << time_3 << " ms" << std::endl;
    hipEventDestroy(start3);
    hipEventDestroy(stop3);


    hipFree(d_A_int);
    hipFree(d_B_int);
    hipFree(d_C_int);
    hipFree(d_A_float);
    hipFree(d_B_float);
    hipFree(d_C_float);
    hipFree(d_A_double);
    hipFree(d_B_double);
    hipFree(d_C_double);

    return 0;
}
